#include "hip/hip_runtime.h"
#include "../CudaCheckError/cudacheckerror.cuh"
#include "../CudaTimer/cudatimer.cuh"
#include "../CudaKernelsHostDeviceFunctions/KernelsHostDeviceFunctions.cuh"

extern "C"{

#include "../CudaCure.h"

}

extern "C"
{
void GPU_initialization_phase(struct BTree *item,struct nnb_info *GPUnnb,int npat,int rsize,int norm,int lpat,double *kernel_timer)
{

struct BTree *GPUitem;
struct nnb_info *KERNEL_GPUnnb;

CudaErrors(hipMalloc((void**)&GPUitem,npat*sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));

DATATYPE **rep;
DATATYPE *rep1;

for(unsigned int i=0;i<npat;i++)
{
CudaErrors(hipMalloc((void**)&rep,item[i].size*sizeof(DATATYPE*)));

for(unsigned int j=0;j<item[i].size;j++)
{  
CudaErrors(hipMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));   
CudaErrors(hipMemcpy(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice)); 
CudaErrors(hipMemcpy(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice)); 
}

CudaErrors(hipMemcpy(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice));
}

transfer_data_to_constant_memory(lpat,rsize,norm);

CudaErrors(hipMalloc((void**)&KERNEL_GPUnnb,npat*sizeof(nnb_info)));

gridsize=(npat+blocksize-1)/blocksize;

CudaTimer timer;

CreateTimer(&timer);
StartTimer(&timer,0);   

GPU_initialization_phase1<<<gridsize,blocksize>>>(GPUitem,KERNEL_GPUnnb,npat);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(GPUnnb,KERNEL_GPUnnb,npat*sizeof(nnb_info),hipMemcpyDeviceToHost));

CudaErrors(hipFree(rep1));
CudaErrors(hipFree(rep));
CudaErrors(hipFree(GPUitem));

CudaErrors(hipFree(KERNEL_GPUnnb));
}

void GPU_minimum_distance_pair(struct BTree *item,struct nnb_info *GPUnnb,int npat,int *gpair1,double *gmin_dist,double *kernel_timer)
{
struct BTree *GPUitem;
struct nnb_info *KERNEL_GPUnnb;

int *block_index_minimum;
int *lpair1;
int *GPU_counter;

int counter=0;

double *block_distance_minimum;
double *lmin_dist;

CudaTimer timer;

CudaErrors(hipMalloc(&GPUitem,npat*sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&KERNEL_GPUnnb,npat*sizeof(nnb_info)));
CudaErrors(hipMemcpy(KERNEL_GPUnnb,GPUnnb,npat*sizeof(nnb_info),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&lpair1,npat*sizeof(int)));
CudaErrors(hipMalloc((void**)&lmin_dist,npat*sizeof(double)));

CudaErrors(hipMalloc((void**)&GPU_counter,sizeof(int)));

gridsize=(npat+blocksize-1)/blocksize;

CudaErrors(hipMalloc((void**)&block_index_minimum,gridsize*sizeof(int)));
CudaErrors(hipMalloc((void**)&block_distance_minimum,gridsize*sizeof(double)));

CreateTimer(&timer);
StartTimer(&timer,0);

NO_NONE_MDP<<<gridsize,blocksize>>>(GPUitem,KERNEL_GPUnnb,lpair1,lmin_dist,GPU_counter,npat);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(&counter,GPU_counter,sizeof(int),hipMemcpyDeviceToHost));

gridsize=(counter+blocksize-1)/blocksize;

CreateTimer(&timer);

StartTimer(&timer,0);   

GPU_find_minimum_distance_pair_per_block<<<gridsize,blocksize>>>(lpair1,lmin_dist,block_index_minimum,block_distance_minimum,counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipFree(lpair1));
CudaErrors(hipFree(lmin_dist));

if(gridsize>1)
{
CudaErrors(hipFree(GPUitem));
CudaErrors(hipFree(KERNEL_GPUnnb));

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0); 

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(block_index_minimum,block_distance_minimum,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);

*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;
}
   
}else
{
CudaErrors(hipFree(GPUitem));
CudaErrors(hipFree(KERNEL_GPUnnb));
}

CudaErrors(hipMemcpy(gpair1,block_index_minimum,sizeof(int),hipMemcpyDeviceToHost));
CudaErrors(hipMemcpy(gmin_dist,block_distance_minimum,sizeof(double),hipMemcpyDeviceToHost));

CudaErrors(hipFree(block_index_minimum));
CudaErrors(hipFree(block_distance_minimum));
}

void GPU_update_nnb(struct BTree *item,struct nnb_info *GPUnnb,int npat,int rsize,int norm,int lpat,int pair1,int pair2,int *gpair1,double *gmin_dist,double *kernel_timer)
{
struct BTree *GPUitem;
struct nnb_info *KERNEL_GPUnnb;

int *block_index_minimum;
int *lpair1;
int *GPU_counter;
int *indices;
int *temp_indices;

int counter=0;
int counter1=0;
int node;

double *block_distance_minimum;
double *lmin_dist;
double *temp_distances;

hipStream_t streams[3];

for(unsigned int i=0; i<3; i++)
{
CudaErrors(hipStreamCreate(&streams[i]));
}

CudaErrors(hipMalloc((void**)&lpair1,npat*sizeof(int)));
CudaErrors(hipMalloc((void**)&indices,npat*sizeof(int)));
CudaErrors(hipMalloc((void**)&lmin_dist,npat*sizeof(double)));  

CudaErrors(hipMalloc((void**)&GPUitem,npat*sizeof(BTree)));

CudaErrors(hipMemcpyAsync(GPUitem,item,pair2*sizeof(BTree),hipMemcpyHostToDevice,streams[0]));

DATATYPE **rep;
DATATYPE *rep1;

for(unsigned int i=0;i<pair2;i++)
{

unsigned int maxrep=(item[i].size<=rsize)?item[i].size:rsize;

CudaErrors(hipMalloc((void**)&rep,maxrep*sizeof(DATATYPE*)));

for(unsigned int j=0;j<maxrep;j++)
{

CudaErrors(hipMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));

CudaErrors(hipMemcpyAsync(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice,streams[0])); 
CudaErrors(hipMemcpyAsync(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice,streams[0])); 
}

CudaErrors(hipMemcpyAsync(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice,streams[0]));

} 

CudaErrors(hipMemcpyAsync(GPUitem+pair2,item+pair2,(pair1-pair2)*sizeof(BTree),hipMemcpyHostToDevice,streams[1]));

for(unsigned int i=pair2;i<pair1;i++)
{


unsigned int maxrep=(item[i].size<=rsize)?item[i].size:rsize;

CudaErrors(hipMalloc((void**)&rep,maxrep*sizeof(DATATYPE*)));

for(unsigned int j=0;j<maxrep;j++)
{

CudaErrors(hipMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));

CudaErrors(hipMemcpyAsync(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice,streams[1])); 
CudaErrors(hipMemcpyAsync(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice,streams[1]));  
}

CudaErrors(hipMemcpyAsync(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice,streams[1]));

} 

CudaErrors(hipMemcpyAsync(GPUitem+pair1,item+pair1,(npat-pair1)*sizeof(BTree),hipMemcpyHostToDevice,streams[2]));

for(unsigned int i=pair1;i<npat;i++)
{

unsigned int maxrep=(item[i].size<=rsize)?item[i].size:rsize;

CudaErrors(hipMalloc((void**)&rep,maxrep*sizeof(DATATYPE*)));

for(unsigned int j=0;j<maxrep;j++)
{

CudaErrors(hipMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));

CudaErrors(hipMemcpyAsync(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice,streams[2])); 
CudaErrors(hipMemcpyAsync(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice,streams[2]));
}

CudaErrors(hipMemcpyAsync(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice,streams[2]));

} 

CudaErrors(hipMalloc((void**)&KERNEL_GPUnnb,npat*sizeof(nnb_info)));

CudaErrors(hipMemcpyAsync(KERNEL_GPUnnb,GPUnnb,pair2*sizeof(nnb_info),hipMemcpyHostToDevice,streams[0]));
CudaErrors(hipMemcpyAsync(KERNEL_GPUnnb+pair2,GPUnnb+pair2,(pair1-pair2)*sizeof(nnb_info),hipMemcpyHostToDevice,streams[1]));
CudaErrors(hipMemcpyAsync(KERNEL_GPUnnb+pair1,GPUnnb+pair1,(npat-pair1)*sizeof(nnb_info),hipMemcpyHostToDevice,streams[2]));

CudaErrors(hipMalloc((void**)&GPU_counter,sizeof(int)));
CudaErrors(hipMemcpy(GPU_counter,&counter,sizeof(int),hipMemcpyHostToDevice));

gridsize=(npat+blocksize-1)/blocksize;

CudaErrors(hipMalloc((void**)&block_index_minimum,gridsize*sizeof(int)));
CudaErrors(hipMalloc((void**)&block_distance_minimum,gridsize*sizeof(double)));

CudaErrors(hipMalloc((void**)&temp_indices,gridsize*sizeof(int)));
CudaErrors(hipMalloc((void**)&temp_distances,gridsize*sizeof(double)));

CudaTimer timer;

CreateTimer(&timer);
StartTimer(&timer,streams[0]);
GPU_update_nnb_kernel<<<(pair2+blocksize-1)/blocksize,blocksize,0,streams[0]>>>(GPUitem,KERNEL_GPUnnb,pair2,lpair1,lmin_dist);
CudaErrors(hipStreamSynchronize(streams[0]));
StopTimer(&timer,streams[0]);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[1]);
GPU_update_nnb_kernel1<<<((pair1-pair2)+blocksize-1)/blocksize,blocksize,0,streams[1]>>>(GPUitem,KERNEL_GPUnnb,pair1,pair1,pair2,lpair1,lmin_dist,pair2);
CudaErrors(hipStreamSynchronize(streams[1]));
StopTimer(&timer,streams[1]);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

transfer_data_to_constant_memory(lpat,rsize,norm);

CreateTimer(&timer);
StartTimer(&timer,streams[2]);
GPU_update_nnb_kernel2<<<((npat-pair1)+blocksize-1)/blocksize,blocksize,0,streams[2]>>>(GPUitem,KERNEL_GPUnnb,npat,pair1,pair2,lpair1,lmin_dist,pair1);
CudaErrors(hipStreamSynchronize(streams[2]));
StopTimer(&timer,streams[2]);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpyAsync(GPUnnb+pair1,KERNEL_GPUnnb+pair1,(npat-pair1)*sizeof(nnb_info),hipMemcpyDeviceToHost,streams[2]));

for(unsigned int i = 0; i<3; i++)
{
CudaErrors(hipStreamDestroy(streams[i]));
}

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_nnb_kernel3<<<((npat-pair2)+blocksize-1)/blocksize,blocksize>>>(GPUitem,KERNEL_GPUnnb,npat,pair1,pair2,pair2,GPU_counter,indices);
CudaErrors(hipDeviceSynchronize());
StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(&counter,GPU_counter,sizeof(int),hipMemcpyDeviceToHost));

for(unsigned int i=0;i<counter;i++)
{

CudaErrors(hipMemcpy(&node,&indices[i],sizeof(int),hipMemcpyDeviceToHost));

counter1=0;
CudaErrors(hipMemcpy(GPU_counter,&counter1,sizeof(int),hipMemcpyHostToDevice)); 

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel<<<(node+blocksize-1)/blocksize+1,blocksize>>>(GPUitem,node,temp_indices,temp_distances,GPU_counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(&counter1,GPU_counter,sizeof(int),hipMemcpyDeviceToHost));

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<((((node+blocksize-1)/blocksize)+1)+blocksize-1)/blocksize+1,blocksize>>>(temp_indices,temp_distances,(node+blocksize-1)/blocksize+1,counter1);
StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

if(((node+blocksize-1)/blocksize+1)>1)
{

if(counter1>0)
{
unsigned int gridsizes5=(node+blocksize-1)/blocksize+1;

if(gridsizes5>counter1)
{
gridsizes5=counter1;
}

while(gridsizes5>1)
{
new_gridsize=(gridsizes5+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel1<<<new_gridsize,blocksize>>>(gridsizes5,temp_indices,temp_distances);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsizes5=new_gridsize;
}

} 
}

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel2<<<1,1>>>(GPUitem,KERNEL_GPUnnb,node,temp_indices,temp_distances,lpair1,lmin_dist);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

}

CudaErrors(hipFree(indices));

CudaErrors(hipMemcpy(GPUnnb,KERNEL_GPUnnb,npat*sizeof(nnb_info),hipMemcpyDeviceToHost));

counter=0;
CudaErrors(hipMemcpy(GPU_counter,&counter,sizeof(int),hipMemcpyHostToDevice));

CreateTimer(&timer);
StartTimer(&timer,0);  
GPU_update_kernel3<<<gridsize,blocksize>>>(lpair1,lmin_dist,block_index_minimum,block_distance_minimum,npat,GPU_counter);
CudaErrors(hipDeviceSynchronize());
StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(&counter,GPU_counter,sizeof(int),hipMemcpyDeviceToHost));

unsigned int gridsizes5=(gridsize+blocksize-1)/blocksize+1;

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<gridsizes5,blocksize>>>(block_index_minimum,block_distance_minimum,gridsize,counter);
CudaErrors(hipDeviceSynchronize());
StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipFree(lpair1));
CudaErrors(hipFree(lmin_dist));

CudaErrors(hipFree(GPU_counter));

if(gridsize>1)
{

CudaErrors(hipFree(rep1));
CudaErrors(hipFree(rep));
CudaErrors(hipFree(GPUitem));

CudaErrors(hipFree(KERNEL_GPUnnb));

if(counter>0)
{

if(gridsize>counter)
{
gridsize=counter;
}

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(block_index_minimum,block_distance_minimum,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;
}

} 

}else

{
CudaErrors(hipFree(rep1));
CudaErrors(hipFree(rep));
CudaErrors(hipFree(GPUitem));

CudaErrors(hipFree(KERNEL_GPUnnb));
}

CudaErrors(hipMemcpy(gpair1,block_index_minimum,sizeof(int),hipMemcpyDeviceToHost));
CudaErrors(hipMemcpy(gmin_dist,block_distance_minimum,sizeof(double),hipMemcpyDeviceToHost));

CudaErrors(hipFree(block_index_minimum));
CudaErrors(hipFree(block_distance_minimum));

}

void GPU_first_pruning(struct BTree *item,int npat,int *nodes,int *pruned_nodes,double *kernel_timer)
{

struct BTree *GPUitem;

int *KERNEL_GPU_nodes;
int *KERNEL_GPU_pruned_nodes;

CudaErrors(hipMalloc((void**)&KERNEL_GPU_nodes,sizeof(int)));
CudaErrors(hipMemcpy(KERNEL_GPU_nodes,nodes,sizeof(int),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&KERNEL_GPU_pruned_nodes,sizeof(int)));
CudaErrors(hipMemcpy(KERNEL_GPU_pruned_nodes,pruned_nodes,sizeof(int),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&GPUitem, npat * sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));

gridsize=(npat+blocksize-1)/blocksize;

CudaTimer timer;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_first_pruning_kernel<<<gridsize,blocksize>>>(GPUitem,npat,KERNEL_GPU_nodes,KERNEL_GPU_pruned_nodes);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(item,GPUitem,npat*sizeof(BTree),hipMemcpyDeviceToHost)); 
CudaErrors(hipMemcpy(nodes,KERNEL_GPU_nodes,sizeof(int),hipMemcpyDeviceToHost));
CudaErrors(hipMemcpy(pruned_nodes,KERNEL_GPU_pruned_nodes,sizeof(int),hipMemcpyDeviceToHost));

CudaErrors(hipFree(GPUitem));
CudaErrors(hipFree(KERNEL_GPU_nodes));
CudaErrors(hipFree(KERNEL_GPU_pruned_nodes));

}

void GPU_second_pruning(struct BTree *item,int npat,int *nodes,int *pruned_nodes,double *kernel_timer)
{

struct BTree *GPUitem;

int *KERNEL_GPU_nodes;
int *KERNEL_GPU_pruned_nodes;

CudaErrors(hipMalloc((void**)&KERNEL_GPU_nodes,sizeof(int)));
CudaErrors(hipMemcpy(KERNEL_GPU_nodes,nodes,sizeof(int),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&KERNEL_GPU_pruned_nodes,sizeof(int)));
CudaErrors(hipMemcpy(KERNEL_GPU_pruned_nodes,pruned_nodes,sizeof(int),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&GPUitem, npat * sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));

gridsize=(npat+blocksize-1)/blocksize;

CudaTimer timer;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_second_pruning_kernel<<<gridsize,blocksize>>>(GPUitem,npat,KERNEL_GPU_nodes,KERNEL_GPU_pruned_nodes);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(item,GPUitem,npat*sizeof(BTree),hipMemcpyDeviceToHost)); 
CudaErrors(hipMemcpy(nodes,KERNEL_GPU_nodes,sizeof(int),hipMemcpyDeviceToHost));
CudaErrors(hipMemcpy(pruned_nodes,KERNEL_GPU_pruned_nodes,sizeof(int),hipMemcpyDeviceToHost));

CudaErrors(hipFree(GPUitem));
CudaErrors(hipFree(KERNEL_GPU_nodes));
CudaErrors(hipFree(KERNEL_GPU_pruned_nodes));

}

void GPU_pruning(struct BTree *item,struct nnb_info *GPUnnb,int npat,int rsize,int norm,int lpat,int *gpair1,double *gmin_dist,double *kernel_timer)
{

struct BTree *GPUitem;

struct nnb_info *KERNEL_GPUnnb;

int *block_index_minimum;
int *GPU_counter;
int *lpair1;
int *indices;

int counter=0;
double *block_distance_minimum;
double *lmin_dist;

CudaErrors(hipMalloc((void**)&lpair1,npat*sizeof(int)));
CudaErrors(hipMalloc((void**)&indices,npat*sizeof(int)));
CudaErrors(hipMalloc((void**)&lmin_dist,npat*sizeof(double)));

hipStream_t stream1,stream2;

hipStreamCreate(&stream1);
hipStreamCreate(&stream2);

CudaErrors(hipMalloc((void**)&GPUitem,npat*sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));  

DATATYPE **rep,*rep1;
for(unsigned int i=0;i<npat;i++)
{

unsigned int maxrep=(item[i].size<=rsize)?item[i].size:rsize;

CudaErrors(hipMalloc((void**)&rep,maxrep*sizeof(DATATYPE*)));

for(unsigned int j=0;j<maxrep;j++)
{

CudaErrors(hipMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));

CudaErrors(hipMemcpy(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice)); 
CudaErrors(hipMemcpy(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice)); 
}

CudaErrors(hipMemcpy(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice));

} 

CudaErrors(hipMalloc((void**)&KERNEL_GPUnnb,npat*sizeof(nnb_info)));
CudaErrors(hipMemcpy(KERNEL_GPUnnb,GPUnnb,npat*sizeof(nnb_info),hipMemcpyHostToDevice));

CudaErrors(hipMalloc((void**)&GPU_counter,sizeof(int)));
CudaErrors(hipMemcpyAsync(GPU_counter,&counter,sizeof(int),hipMemcpyHostToDevice,stream2));

gridsize=(npat+blocksize-1)/blocksize;

CudaErrors(hipMalloc((void**)&block_index_minimum,gridsize*sizeof(int)));
CudaErrors(hipMalloc((void**)&block_distance_minimum,gridsize*sizeof(double)));

CudaTimer timer;

CreateTimer(&timer);
StartTimer(&timer,stream1);        

GPU_update_nnb_kernel<<<gridsize,blocksize,0,stream1>>>(GPUitem,KERNEL_GPUnnb,npat,lpair1,lmin_dist);
CudaErrors(hipStreamSynchronize(stream1));

StopTimer(&timer,stream1);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,stream2);        

GPU_pruning_kernel1<<<gridsize,blocksize,0,stream2>>>(GPUitem,KERNEL_GPUnnb,npat,GPU_counter,indices);
CudaErrors(hipStreamSynchronize(stream2));

StopTimer(&timer,stream2);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpyAsync(&counter,GPU_counter,sizeof(int),hipMemcpyDeviceToHost,stream2));

hipStreamDestroy(stream1);
hipStreamDestroy(stream2);

transfer_data_to_constant_memory(lpat,rsize,norm);

unsigned int gridsizes1=(counter+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0); 

GPU_pruning_kernel2<<<gridsizes1,blocksize>>>(GPUitem,KERNEL_GPUnnb,indices,counter,lpair1,lmin_dist);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

counter=0;
CudaErrors(hipMemcpy(GPU_counter,&counter,sizeof(int),hipMemcpyHostToDevice));

CreateTimer(&timer);
StartTimer(&timer,0);  

GPU_update_kernel3<<<gridsize,blocksize>>>(lpair1,lmin_dist,block_index_minimum,block_distance_minimum,npat,GPU_counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipMemcpy(&counter,GPU_counter,sizeof(int),hipMemcpyDeviceToHost));

unsigned int gridsizes5=(gridsize+blocksize-1)/blocksize+1;

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<gridsizes5,blocksize>>>(block_index_minimum,block_distance_minimum,gridsize,counter);
CudaErrors(hipDeviceSynchronize());
StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CudaErrors(hipFree(lpair1));
CudaErrors(hipFree(lmin_dist));

CudaErrors(hipFree(GPU_counter));

CudaErrors(hipMemcpy(GPUnnb,KERNEL_GPUnnb,npat*sizeof(nnb_info),hipMemcpyDeviceToHost));

if(gridsize>1)
{
CudaErrors(hipFree(rep1));
CudaErrors(hipFree(rep));
CudaErrors(hipFree(GPUitem));

CudaErrors(hipFree(KERNEL_GPUnnb));

if(counter>0) 
{  
if(gridsize>counter)
{
gridsize=counter;
}

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);  


GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(block_index_minimum,block_distance_minimum,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
*kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;
}

}

}else
{
CudaErrors(hipFree(rep1));
CudaErrors(hipFree(rep));
CudaErrors(hipFree(GPUitem));

CudaErrors(hipFree(KERNEL_GPUnnb));
}  

CudaErrors(hipMemcpy(gpair1,block_index_minimum,sizeof(int),hipMemcpyHostToHost));
CudaErrors(hipMemcpy(gmin_dist,block_distance_minimum,sizeof(double),hipMemcpyHostToHost));

CudaErrors(hipFree(block_index_minimum));
CudaErrors(hipFree(block_distance_minimum));

}

void Gpu_Cuda_Cure_Part_Results(int npat,int lpat,int csize,double init_timer,double find_mdp_timer,double clustering_timer,double merge_timer,double update_timer,double pruning_timer,const char *type)
{

struct Timers *BT=(Timers*)malloc(sizeof(Timers));
const char *path;
   
BT->init_timer=Timer_Array(1);
BT->find_mdp_timer=Timer_Array(1);
BT->clustering_timer=Timer_Array(1);
BT->merge_timer=Timer_Array(1);
BT->update_timer=Timer_Array(1);
BT->pruning_timer=Timer_Array(1);

BT->init_timer[0]=init_timer;
BT->find_mdp_timer[0]=find_mdp_timer;
BT->clustering_timer[0]=clustering_timer;
BT->merge_timer[0]=merge_timer;
BT->update_timer[0]=update_timer;
BT->pruning_timer[0]=pruning_timer;

char *previous_path=current_path();

if(strcmp(type,"SerialCure")==0)
{
path="/SerialCure";

}else if(strcmp(type,"CudaCurePart")==0)
{
path="/CudaCurePart";
}

path=ConcenateString(previous_path,path);  
change_directory(path);

show_time_results_for_each_case(type,BT,npat,lpat,csize);

change_directory(previous_path);

}

void Gpu_Cuda_Cure_Print_Results(struct BTree *item,int npat,int rsize,int lpat,int clusters,const char *type)
{

const char *path;
char *previous_path=current_path();

if(strcmp(type,"SerialCure")==0)
{
path="/SerialCure";

}else if(strcmp(type,"CudaCurePart")==0)
{
path="/CudaCurePart";
}

path=ConcenateString(previous_path,path);

change_directory(path);

print_results(type,item,npat,clusters,rsize,lpat);

change_directory(previous_path);

}

}