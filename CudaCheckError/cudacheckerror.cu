#include "cudacheckerror.cuh" 

// Function to check CUDA errors
void CudaErrors(hipError_t cudaStat)
{  
    // Check if the CUDA call was successful
    if (cudaStat != hipSuccess)
    {
        // If there was an error, print the error message to the console
        printf("%s\n", hipGetErrorString(cudaStat));
        
        exit(1);
    }
} 