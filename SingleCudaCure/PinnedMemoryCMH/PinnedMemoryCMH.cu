#include "hip/hip_runtime.h"
#include "../../CudaCheckError/cudacheckerror.cuh"
#include "../../CudaTimer/cudatimer.cuh"
#include "../../CudaKernelsHostDeviceFunctions/KernelsHostDeviceFunctions.cuh"

extern "C"
{
#include "../../CudaCure.h"
}


extern "C"
{

void GPU_Clustering_Pinned_Memory_CMH(struct BTree *item,int npat,int rsize,int norm,int lpat,int csize)
{

printf("--------------------------------------------------------------PINNED MEMORY CMH-------------------------------------------------------------------------\n\n");

CudaTimer timer,cluster_timer;

struct Timers *BT1=(Timers*)malloc(sizeof(Timers));

BT1->init_timer=Timer_Array(1);
BT1->find_mdp_timer=Timer_Array(1);
BT1->clustering_timer=Timer_Array(1);
BT1->merge_timer=Timer_Array(1);
BT1->update_timer=Timer_Array(1);
BT1->pruning_timer=Timer_Array(1);

hipStream_t streams[4];

for(unsigned int i=0; i<4; i++)
{
CudaErrors(hipStreamCreate(&streams[i]));
}

struct BTree *GPUitem;
struct BTree *GPU_merge_tmp_item;

struct nnb_info *KERNEL_GPUnnb;

int pair1,pair2;

int *gpair1;
int *temp_indices;
int *nodes;
int *pruned_nodes;
int *counter;
int *counter1;
int *indices;
int *lpair1;

double *gmin_dist;
double *lmin_dist;
double *min_dist;
double *temp_distances;

gridsize=(npat+blocksize-1)/blocksize;

CudaErrors(hipHostMalloc((void**)&gpair1,gridsize*sizeof(int)));
CudaErrors(hipHostMalloc((void**)&gmin_dist,gridsize*sizeof(double)));

CudaErrors(hipHostMalloc((void**)&temp_indices,gridsize*sizeof(int)));
CudaErrors(hipHostMalloc((void**)&temp_distances,gridsize*sizeof(double)));

CudaErrors(hipHostMalloc((void**)&nodes,sizeof(int)));
CudaErrors(hipHostMalloc((void**)&pruned_nodes,sizeof(int)));
*pruned_nodes=0;
*nodes=npat;

CudaErrors(hipHostMalloc((void**)&counter,sizeof(int)));
CudaErrors(hipHostMalloc((void**)&counter1,sizeof(int)));

CudaErrors(hipHostMalloc((void**)&indices,npat*sizeof(int)));
CudaErrors(hipHostMalloc((void**)&lpair1,npat*sizeof(int)));
CudaErrors(hipHostMalloc((void**)&lmin_dist,npat*sizeof(double)));

CudaErrors(hipHostMalloc((void**)&min_dist,sizeof(double)));

/*-----------------------------------------------------------------------------GPUitem---------------------------------------------------------------------------------*/

CudaErrors(hipHostMalloc((void**)&GPUitem,npat*sizeof(BTree)));
CudaErrors(hipMemcpy(GPUitem,item,npat*sizeof(BTree),hipMemcpyHostToDevice));


int **pats,*pats1;
DATATYPE **rep,*rep1,*mean;

for(unsigned int i=0;i<npat; i++)
{


CudaErrors(hipHostMalloc((void**)&rep,rsize*sizeof(DATATYPE*)));

register int maxrep=(item[i].size<=rsize)?item[i].size:rsize;

for(register int j=0;j<maxrep;j++)
{

CudaErrors(hipHostMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));

CudaErrors(hipMemcpy(rep1,item[i].rep[j],lpat*sizeof(DATATYPE),hipMemcpyHostToDevice)); 
CudaErrors(hipMemcpy(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice)); 
}

if(maxrep<rsize)
{

for(register int j=maxrep;j<rsize;j++)
{

CudaErrors(hipHostMalloc((void**)&rep1,lpat*sizeof(DATATYPE)));
CudaErrors(hipMemcpy(&rep[j],&rep1,sizeof(DATATYPE*),hipMemcpyHostToDevice)); 
}
}  
CudaErrors(hipMemcpy(&(GPUitem[i].rep),&rep,sizeof(DATATYPE**),hipMemcpyHostToDevice));

CudaErrors(hipHostMalloc((void**)&mean,lpat*sizeof(DATATYPE)));
CudaErrors(hipMemcpy(mean,item[i].mean,lpat*sizeof(DATATYPE),hipMemcpyHostToDevice));
CudaErrors(hipMemcpy(&(GPUitem[i].mean),&mean,sizeof(DATATYPE*),hipMemcpyHostToDevice));

}

CudaErrors(hipHostMalloc((void**)&pats,npat*sizeof(int*)));
for(unsigned int i=0;i<npat;i++)
{
CudaErrors(hipHostMalloc((void**)&pats1,nnpc*sizeof(int)));   
CudaErrors(hipMemcpy(pats1,item[i].pats,nnpc*sizeof(int),hipMemcpyHostToDevice));
CudaErrors(hipMemcpy(&(pats[i]),&pats1,sizeof(int*),hipMemcpyHostToDevice));
}


/*-----------------------------------------------------------------------GPU_merge_tmp_item-----------------------------------------------------------------------*/

CudaErrors(hipHostMalloc((void**)&GPU_merge_tmp_item,sizeof(BTree)));

DATATYPE **rep2,*rep3;
CudaErrors(hipHostMalloc((void**)&rep2,(2*rsize)*sizeof(DATATYPE*)));

for(unsigned int i =0; i<(2*rsize); i++)
{
CudaErrors(hipHostMalloc((void**)&rep3,lpat*sizeof(DATATYPE)));  
CudaErrors(hipMemcpy(&rep2[i],&rep3,sizeof(DATATYPE*),hipMemcpyHostToDevice)); 
}

CudaErrors(hipMemcpy(&(GPU_merge_tmp_item[0].rep),&rep2,sizeof(DATATYPE*),hipMemcpyHostToDevice));


/*-----------------------------------------------------------------------KERNEL_GPUnnb-----------------------------------------------------------------------------------*/
CudaErrors(hipHostMalloc((void**)&KERNEL_GPUnnb,npat*sizeof(nnb_info)));

/*-------------------------------------------------------constant memory-----------------------------------------------------------------------------*/
one_minus_alpha_rev=one_minus_alpha_rev_value();
transfer_data_to_constant_memory(alpha,one_minus_alpha_rev);
transfer_data_to_constant_memory(lpat,rsize,norm);     

/*--------------------------------------------------------------------------------------------------------------------------------------------------------*/

/*------------------------------------------------initialization phase-------------------------------------------------------------------*/



CreateTimer(&timer);
StartTimer(&timer,0);

GPU_initialization_phase1<<<gridsize,blocksize>>>(GPUitem,KERNEL_GPUnnb,npat);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->init_timer[0]=GetElapsedTime(&timer);
DestroyTimer(&timer);



/*------------------------------------------------------------------------------------------------------------------------------------------*/

/*--------------------------------------------------minimum distance pair--------------------------------------------------------------------*/



CreateTimer(&timer);
StartTimer(&timer,0);

NO_NONE_MDP<<<gridsize,blocksize>>>(GPUitem,KERNEL_GPUnnb,lpair1,lmin_dist,counter,npat);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->find_mdp_timer[0]=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=(*counter+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_find_minimum_distance_pair_per_block<<<gridsize,blocksize>>>(lpair1,lmin_dist,gpair1,gmin_dist,*counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->find_mdp_timer[0]+GetElapsedTime(&timer);
DestroyTimer(&timer);

if(gridsize>1)
{
while(gridsize>1)
{

new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(gpair1,gmin_dist,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->find_mdp_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;

}
}





/*--------------------------------------------------------------------------------------------------------------------------------------------*/

printf("ENTERING CLUSTERING PHASE (nodes=%d, clusters=%d)\n", *nodes, csize); fflush(0);

CreateTimer(&cluster_timer);
StartTimer(&cluster_timer,0);

while(*nodes>csize)
{

pair1=gpair1[0];

if(pair1==NONE) break;

gridsize=(npat+blocksize-1)/blocksize;

pair2=KERNEL_GPUnnb[pair1].index;

GPUroot<<<1,1>>>(min_dist,gmin_dist);
CudaErrors(hipDeviceSynchronize());


/*--------------------------------------------------------merge-------------------------------------------------------------------------------------------*/




CreateTimer(&timer);
StartTimer(&timer,0);
GPUmerge<<<1,1>>>(GPUitem,GPU_merge_tmp_item,pats,pair1,pair2,min_dist,size_limit);
CudaErrors(hipDeviceSynchronize());
StopTimer(&timer,0);
BT1->merge_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

/*----------------------------------------------------------------------------------------------------------------------------------------------------------*/  

/*-------------------------------------------------------update nnb----------------------------------------------------------------------------------------------*/

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,streams[0]);
GPU_update_nnb_kernel<<<(pair2+blocksize-1)/blocksize,blocksize,0,streams[0]>>>(GPUitem,KERNEL_GPUnnb,pair2,lpair1,lmin_dist);
CudaErrors(hipStreamSynchronize(streams[0]));
StopTimer(&timer,streams[0]);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[1]);
GPU_update_nnb_kernel1<<<((pair1-pair2)+blocksize-1)/blocksize,blocksize,0,streams[1]>>>(GPUitem,KERNEL_GPUnnb,pair1,pair1,pair2,lpair1,lmin_dist,pair2);
CudaErrors(hipStreamSynchronize(streams[1]));
StopTimer(&timer,streams[1]);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[2]);
GPU_update_nnb_kernel2<<<((npat-pair1)+blocksize-1)/blocksize,blocksize,0,streams[2]>>>(GPUitem,KERNEL_GPUnnb,npat,pair1,pair2,lpair1,lmin_dist,pair1);
CudaErrors(hipStreamSynchronize(streams[2]));
StopTimer(&timer,streams[2]);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[3]);
GPU_update_nnb_kernel3<<<((npat-pair2)+blocksize-1)/blocksize,blocksize,0,streams[3]>>>(GPUitem,KERNEL_GPUnnb,npat,pair1,pair2,pair2,counter,indices);
CudaErrors(hipStreamSynchronize(streams[3]));
StopTimer(&timer,streams[3]);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

for(unsigned int i=0;i<*counter;i++)
{

*counter1=0;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel<<<(indices[i]+blocksize-1)/blocksize+1,blocksize>>>(GPUitem,indices[i],temp_indices,temp_distances,counter1);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);


CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<((((indices[i]+blocksize-1)/blocksize)+1)+blocksize-1)/blocksize+1,blocksize>>>(temp_indices,temp_distances,(indices[i]+blocksize-1)/blocksize+1,*counter1);
StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

if(((indices[i]+blocksize-1)/blocksize+1)>1)
{

if(*counter1>0)
{
unsigned int gridsizes5=(indices[i]+blocksize-1)/blocksize+1;

if(gridsizes5>*counter1)
{
gridsizes5=*counter1;
}

while(gridsizes5>1)
{
new_gridsize=(gridsizes5+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel1<<<new_gridsize,blocksize>>>(gridsizes5,temp_indices,temp_distances);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsizes5=new_gridsize;

}

} 
}


CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel2<<<1,1>>>(GPUitem,KERNEL_GPUnnb,indices[i],temp_indices,temp_distances,lpair1,lmin_dist);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

}

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_update_kernel3<<<gridsize,blocksize>>>(lpair1,lmin_dist,gpair1,gmin_dist,npat,counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<(gridsize+blocksize-1)/blocksize+1,blocksize>>>(gpair1,gmin_dist,gridsize,*counter);
StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);

if((npat+blocksize-1)/blocksize>1)
{

if(*counter>0)
{
unsigned int gridsize = (npat+blocksize-1)/blocksize;
if(gridsize>*counter)
{
gridsize=*counter;
}

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(gpair1,gmin_dist,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
BT1->update_timer[0]+=GetElapsedTime(&timer);
DestroyTimer(&timer);      

gridsize=new_gridsize;
}

} 

}




/*--------------------------------------------------------------------------------------------------------------------------------------------------------------*/


--(*nodes);

/*---------------------------------------------------------------- prune clusters--------------------------------------------------------------------------------*/   
if(prune_clusters)
{

double kernel_timer;

if (*nodes == (int)(npat * FirstPruneRatio)) 
{

kernel_timer=0;

gridsize=(npat+blocksize-1)/blocksize;

printf("==== First phase of pruning at %d nodes remaining ====\n", *nodes);

CreateTimer(&timer);
StartTimer(&timer,0);

GPU_first_pruning_kernel<<<gridsize,blocksize>>>(GPUitem,npat,nodes,pruned_nodes);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer=GetElapsedTime(&timer);
BT1->pruning_timer[0]+=kernel_timer;
DestroyTimer(&timer);

printf("GPU first pruning - (1): %lf seconds (pruned nodes = %d)\n",kernel_timer, *pruned_nodes);

kernel_timer=0;

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,streams[0]);

GPU_update_nnb_kernel<<<gridsize,blocksize,0,streams[0]>>>(GPUitem,KERNEL_GPUnnb,npat,lpair1,lmin_dist);
CudaErrors(hipStreamSynchronize(streams[0]));

StopTimer(&timer,streams[0]);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[1]);

GPU_pruning_kernel1<<<gridsize,blocksize,0,streams[1]>>>(GPUitem,KERNEL_GPUnnb,npat,counter,indices);
CudaErrors(hipStreamSynchronize(streams[1]));

StopTimer(&timer,streams[1]);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,0);  

GPU_pruning_kernel2<<<(*counter+blocksize-1)/blocksize,blocksize>>>(GPUitem,KERNEL_GPUnnb,indices,*counter,lpair1,lmin_dist);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer); 

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,0);   

GPU_update_kernel3<<<gridsize,blocksize>>>(lpair1,lmin_dist,gpair1,gmin_dist,npat,counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);   

unsigned int gridsizes5=(gridsize+blocksize-1)/blocksize+1;

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<gridsizes5,blocksize>>>(gpair1,gmin_dist,gridsize,*counter);
StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer); 

if(gridsize>1)
{


if(*counter>0) 
{  
if(gridsize>*counter)
{
gridsize=*counter;
}

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);            

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(gpair1,gmin_dist,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;
}


}

}

printf("GPU first pruning - (2): %lf seconds\n",kernel_timer);

BT1->pruning_timer[0]+=kernel_timer;





}else if(*nodes == csize * SecondPruneMulti)
{
kernel_timer=0;

gridsize=(npat+blocksize-1)/blocksize;

printf("==== Second phase of pruning at %d nodes remaining ====\n", *nodes);

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_second_pruning_kernel<<<gridsize,blocksize>>>(GPUitem,npat,nodes,pruned_nodes);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer=GetElapsedTime(&timer);
BT1->pruning_timer[0]+=kernel_timer;
DestroyTimer(&timer);

printf("GPU second pruning - (1): %lf seconds - pruned nodes = %d\n",kernel_timer, *pruned_nodes);

kernel_timer=0;

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,streams[0]);

GPU_update_nnb_kernel<<<gridsize,blocksize,0,streams[0]>>>(GPUitem,KERNEL_GPUnnb,npat,lpair1,lmin_dist);
CudaErrors(hipStreamSynchronize(streams[0]));

StopTimer(&timer,streams[0]);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

CreateTimer(&timer);
StartTimer(&timer,streams[1]);

GPU_pruning_kernel1<<<gridsize,blocksize,0,streams[1]>>>(GPUitem,KERNEL_GPUnnb,npat,counter,indices);
CudaErrors(hipStreamSynchronize(streams[1]));

StopTimer(&timer,streams[1]);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);


CreateTimer(&timer);
StartTimer(&timer,0);  

GPU_pruning_kernel2<<<(*counter+blocksize-1)/blocksize,blocksize>>>(GPUitem,KERNEL_GPUnnb,indices,*counter,lpair1,lmin_dist);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer); 

*counter=0;

CreateTimer(&timer);
StartTimer(&timer,0);   

GPU_update_kernel3<<<gridsize,blocksize>>>(lpair1,lmin_dist,gpair1,gmin_dist,npat,counter);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);  

unsigned int gridsizes5=(gridsize+blocksize-1)/blocksize+1;

CreateTimer(&timer);
StartTimer(&timer,0);
GPU_update_kernel4<<<gridsizes5,blocksize>>>(gpair1,gmin_dist,gridsize,*counter);
StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);  

if(gridsize>1)
{


if(*counter>0) 
{  
if(gridsize>*counter)
{
gridsize=*counter;
}

while(gridsize>1)
{
new_gridsize=(gridsize+blocksize-1)/blocksize;

CreateTimer(&timer);
StartTimer(&timer,0);            

GPU_find_minimum_distance_pair<<<new_gridsize,blocksize>>>(gpair1,gmin_dist,gridsize);
CudaErrors(hipDeviceSynchronize());

StopTimer(&timer,0);
kernel_timer+=GetElapsedTime(&timer);
DestroyTimer(&timer);

gridsize=new_gridsize;
}


}

}

printf("GPU second pruning - (2): %lf seconds\n",kernel_timer);

BT1->pruning_timer[0]+=kernel_timer;



}

}



/*------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/   

}

StopTimer(&cluster_timer,0);
BT1->clustering_timer[0]=GetElapsedTime(&cluster_timer);
DestroyTimer(&cluster_timer);

for(unsigned int i=0;i<npat;i++)
{

CudaErrors(hipMemcpy(&(GPUitem[i].pats),&pats[i],sizeof(int*),hipMemcpyHostToDevice)); 
}

char *previous_path=current_path();
const char *path="/SingleCudaCure/PinnedMemoryCMH";
path=ConcenateString(previous_path,path);

change_directory(path);
show_time_results_for_each_case("PinnedMemoryCMH",BT1,npat,lpat,csize);

if(print_clusters)
{
print_results("PinnedMemoryCMH",GPUitem,npat,*nodes,rsize,lpat);
}

change_directory(previous_path);

TimersFree(BT1);

CudaErrors(hipHostFree(indices));
CudaErrors(hipHostFree(pruned_nodes));
CudaErrors(hipHostFree(nodes));
CudaErrors(hipHostFree(lpair1));
CudaErrors(hipHostFree(lmin_dist));
CudaErrors(hipHostFree(counter));
CudaErrors(hipHostFree(counter1));
CudaErrors(hipHostFree(gpair1));
CudaErrors(hipHostFree(gmin_dist));
CudaErrors(hipHostFree(temp_indices));
CudaErrors(hipHostFree(temp_distances));

for(unsigned int i = 0; i<4; i++)
{
CudaErrors(hipStreamDestroy(streams[i]));
}
CudaErrors(hipHostFree(KERNEL_GPUnnb));

CudaErrors(hipHostFree(rep2));
CudaErrors(hipHostFree(rep3));
CudaErrors(hipHostFree(GPU_merge_tmp_item));

CudaErrors(hipHostFree(pats1));
CudaErrors(hipHostFree(pats));
CudaErrors(hipHostFree(mean));

CudaErrors(hipHostFree(rep1));
CudaErrors(hipHostFree(rep));
CudaErrors(hipHostFree(GPUitem));


}


}