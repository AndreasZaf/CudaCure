#include "cudatimer.cuh"

// Function to create and initialize the CUDA timer events
void CreateTimer(CudaTimer* Timer)
{
    // Create a CUDA event for the start timer
    CudaErrors(hipEventCreate(&Timer->Starttimer));
    
    // Create a CUDA event for the end timer
    CudaErrors(hipEventCreate(&Timer->Endtimer));
}

// Function to destroy the CUDA timer events and free resources
void DestroyTimer(CudaTimer* Timer)
{
    // Destroy the start timer event
    CudaErrors(hipEventDestroy(Timer->Starttimer));
    
    // Destroy the end timer event
    CudaErrors(hipEventDestroy(Timer->Endtimer));
}

// Function to start the timer using the specified CUDA stream
void StartTimer(CudaTimer* Timer, hipStream_t stream)
{
    // If no stream is specified (stream == 0), record the start event without a stream
    if (stream == 0)
    {
        CudaErrors(hipEventRecord(Timer->Starttimer));
    }
    else
    {
        // Record the start event on the specified stream
        CudaErrors(hipEventRecord(Timer->Starttimer, stream));
    }
}

// Function to stop the timer using the specified CUDA stream
void StopTimer(CudaTimer* Timer, hipStream_t stream)
{
    // If no stream is specified (stream == 0), record the end event without a stream
    if (stream == 0)
    {
        CudaErrors(hipEventRecord(Timer->Endtimer));
    }
    else
    {
        // Record the end event on the specified stream
        CudaErrors(hipEventRecord(Timer->Endtimer, stream));
    }
    
    // Wait for the end event to complete before proceeding
    CudaErrors(hipEventSynchronize(Timer->Endtimer));
}

// Function to retrieve the elapsed time in seconds between the start and stop events
double GetElapsedTime(CudaTimer* Timer)
{
    float elapsedTime;

    // Calculate the elapsed time in milliseconds between start and end events
    CudaErrors(hipEventElapsedTime(&elapsedTime, Timer->Starttimer, Timer->Endtimer));
    
    // Convert milliseconds to seconds and return the result
    return (double)elapsedTime / 1000.0;
}